#include "hip/hip_runtime.h"
/* r1mach.f -- translated by f2c (version 20100827).
   You must link the resulting object file with libf2c:
	on Microsoft Windows system, link with libf2c.lib;
	on Linux or Unix systems, link with .../path/to/libf2c.a -lm
	or, if you install libf2c.a in a standard place, with -lf2c -lm
	-- in that order, at the end of the command line, as in
		cc *.o -lf2c -lm
	Source for libf2c is in /netlib/f2c/libf2c.zip, e.g.,

		http://www.netlib.org/f2c/libf2c.zip
*/

#include "f2c.h"

/* Table of constant values */

extern __device__ integer c__1;
extern __device__ integer c__2;

__device__ __constant__ real equiv_4[5] = {
  1.18E-38,
  3.40E+38,
  0.595E-07,
  1.19E-07,
  0.30102999566
};


/* DECK R1MACH */
__device__ doublereal r1mach_(const integer *i__)
{
    /* System generated locals */
    real ret_val;

/*     /\* Local variables *\/ */
/* #define log10 ((integer *)equiv_4 + 4) */
/* #define large ((integer *)equiv_4 + 1) */
#define rmach (equiv_4)
/* #define small ((integer *)equiv_4) */
/* #define diver ((integer *)equiv_4 + 3) */
/* #define right ((integer *)equiv_4 + 2) */


    __device__ extern /* Subroutine */ int xermsg_(char *, char *, char *, const integer *, 
	    const integer *, ftnlen, ftnlen, ftnlen);

/* ***BEGIN PROLOGUE  R1MACH */
/* ***PURPOSE  Return floating point machine dependent constants. */
/* ***LIBRARY   SLATEC */
/* ***CATEGORY  R1 */
/* ***TYPE      SINGLE PRECISION (R1MACH-S, D1MACH-D) */
/* ***KEYWORDS  MACHINE CONSTANTS */
/* ***AUTHOR  Fox, P. A., (Bell Labs) */
/*           Hall, A. D., (Bell Labs) */
/*           Schryer, N. L., (Bell Labs) */
/* ***DESCRIPTION */

/*   R1MACH can be used to obtain machine-dependent parameters for the */
/*   local machine environment.  It is a function subprogram with one */
/*   (input) argument, and can be referenced as follows: */

/*        A = R1MACH(I) */

/*   where I=1,...,5.  The (output) value of A above is determined by */
/*   the (input) value of I.  The results for various values of I are */
/*   discussed below. */

/*   R1MACH(1) = B**(EMIN-1), the smallest positive magnitude. */
/*   R1MACH(2) = B**EMAX*(1 - B**(-T)), the largest magnitude. */
/*   R1MACH(3) = B**(-T), the smallest relative spacing. */
/*   R1MACH(4) = B**(1-T), the largest relative spacing. */
/*   R1MACH(5) = LOG10(B) */

/*   Assume single precision numbers are represented in the T-digit, */
/*   base-B form */

/*              sign (B**E)*( (X(1)/B) + ... + (X(T)/B**T) ) */

/*   where 0 .LE. X(I) .LT. B for I=1,...,T, 0 .LT. X(1), and */
/*   EMIN .LE. E .LE. EMAX. */

/*   The values of B, T, EMIN and EMAX are provided in I1MACH as */
/*   follows: */
/*   I1MACH(10) = B, the base. */
/*   I1MACH(11) = T, the number of base-B digits. */
/*   I1MACH(12) = EMIN, the smallest exponent E. */
/*   I1MACH(13) = EMAX, the largest exponent E. */

/*   To alter this function for a particular environment, the desired */
/*   set of DATA statements should be activated by removing the C from */
/*   column 1.  Also, the values of R1MACH(1) - R1MACH(4) should be */
/*   checked for consistency with the local operating system. */

/* ***REFERENCES  P. A. Fox, A. D. Hall and N. L. Schryer, Framework for */
/*                 a portable library, ACM Transactions on Mathematical */
/*                 Software 4, 2 (June 1978), pp. 177-188. */
/* ***ROUTINES CALLED  XERMSG */
/* ***REVISION HISTORY  (YYMMDD) */
/*   790101  DATE WRITTEN */
/*   890213  REVISION DATE from Version 3.2 */
/*   891214  Prologue converted to Version 4.0 format.  (BAB) */
/*   900315  CALLs to XERROR changed to CALLs to XERMSG.  (THJ) */
/*   900618  Added DEC RISC constants.  (WRB) */
/*   900723  Added IBM RS 6000 constants.  (WRB) */
/*   910710  Added HP 730 constants.  (SMR) */
/*   911114  Added Convex IEEE constants.  (WRB) */
/*   920121  Added SUN -r8 compiler option constants.  (WRB) */
/*   920229  Added Touchstone Delta i860 constants.  (WRB) */
/*   920501  Reformatted the REFERENCES section.  (WRB) */
/*   920625  Added CONVEX -p8 and -pd8 compiler option constants. */
/*           (BKS, WRB) */
/*   930201  Added DEC Alpha and SGI constants.  (RWC and WRB) */
/* ***END PROLOGUE  R1MACH */




/*     MACHINE CONSTANTS FOR THE AMIGA */
/*     ABSOFT FORTRAN COMPILER USING THE 68020/68881 COMPILER OPTION */

/*     DATA SMALL(1) / Z'00800000' / */
/*     DATA LARGE(1) / Z'7F7FFFFF' / */
/*     DATA RIGHT(1) / Z'33800000' / */
/*     DATA DIVER(1) / Z'34000000' / */
/*     DATA LOG10(1) / Z'3E9A209B' / */

/*     MACHINE CONSTANTS FOR THE AMIGA */
/*     ABSOFT FORTRAN COMPILER USING SOFTWARE FLOATING POINT */

/*     DATA SMALL(1) / Z'00800000' / */
/*     DATA LARGE(1) / Z'7EFFFFFF' / */
/*     DATA RIGHT(1) / Z'33800000' / */
/*     DATA DIVER(1) / Z'34000000' / */
/*     DATA LOG10(1) / Z'3E9A209B' / */

/*     MACHINE CONSTANTS FOR THE APOLLO */

/*     DATA SMALL(1) / 16#00800000 / */
/*     DATA LARGE(1) / 16#7FFFFFFF / */
/*     DATA RIGHT(1) / 16#33800000 / */
/*     DATA DIVER(1) / 16#34000000 / */
/*     DATA LOG10(1) / 16#3E9A209B / */

/*     MACHINE CONSTANTS FOR THE BURROUGHS 1700 SYSTEM */

/*     DATA RMACH(1) / Z400800000 / */
/*     DATA RMACH(2) / Z5FFFFFFFF / */
/*     DATA RMACH(3) / Z4E9800000 / */
/*     DATA RMACH(4) / Z4EA800000 / */
/*     DATA RMACH(5) / Z500E730E8 / */

/*     MACHINE CONSTANTS FOR THE BURROUGHS 5700/6700/7700 SYSTEMS */

/*     DATA RMACH(1) / O1771000000000000 / */
/*     DATA RMACH(2) / O0777777777777777 / */
/*     DATA RMACH(3) / O1311000000000000 / */
/*     DATA RMACH(4) / O1301000000000000 / */
/*     DATA RMACH(5) / O1157163034761675 / */

/*     MACHINE CONSTANTS FOR THE CDC 170/180 SERIES USING NOS/VE */

/*     DATA RMACH(1) / Z"3001800000000000" / */
/*     DATA RMACH(2) / Z"4FFEFFFFFFFFFFFE" / */
/*     DATA RMACH(3) / Z"3FD2800000000000" / */
/*     DATA RMACH(4) / Z"3FD3800000000000" / */
/*     DATA RMACH(5) / Z"3FFF9A209A84FBCF" / */

/*     MACHINE CONSTANTS FOR THE CDC 6000/7000 SERIES */

/*     DATA RMACH(1) / 00564000000000000000B / */
/*     DATA RMACH(2) / 37767777777777777776B / */
/*     DATA RMACH(3) / 16414000000000000000B / */
/*     DATA RMACH(4) / 16424000000000000000B / */
/*     DATA RMACH(5) / 17164642023241175720B / */

/*     MACHINE CONSTANTS FOR THE CELERITY C1260 */

/*     DATA SMALL(1) / Z'00800000' / */
/*     DATA LARGE(1) / Z'7F7FFFFF' / */
/*     DATA RIGHT(1) / Z'33800000' / */
/*     DATA DIVER(1) / Z'34000000' / */
/*     DATA LOG10(1) / Z'3E9A209B' / */

/*     MACHINE CONSTANTS FOR THE CONVEX */
/*     USING THE -fn COMPILER OPTION */

/*     DATA RMACH(1) / Z'00800000' / */
/*     DATA RMACH(2) / Z'7FFFFFFF' / */
/*     DATA RMACH(3) / Z'34800000' / */
/*     DATA RMACH(4) / Z'35000000' / */
/*     DATA RMACH(5) / Z'3F9A209B' / */

/*     MACHINE CONSTANTS FOR THE CONVEX */
/*     USING THE -fi COMPILER OPTION */

/*     DATA RMACH(1) / Z'00800000' / */
/*     DATA RMACH(2) / Z'7F7FFFFF' / */
/*     DATA RMACH(3) / Z'33800000' / */
/*     DATA RMACH(4) / Z'34000000' / */
/*     DATA RMACH(5) / Z'3E9A209B' / */

/*     MACHINE CONSTANTS FOR THE CONVEX */
/*     USING THE -p8 OR -pd8 COMPILER OPTION */

/*     DATA RMACH(1) / Z'0010000000000000' / */
/*     DATA RMACH(2) / Z'7FFFFFFFFFFFFFFF' / */
/*     DATA RMACH(3) / Z'3CC0000000000000' / */
/*     DATA RMACH(4) / Z'3CD0000000000000' / */
/*     DATA RMACH(5) / Z'3FF34413509F79FF' / */

/*     MACHINE CONSTANTS FOR THE CRAY */

/*     DATA RMACH(1) / 200034000000000000000B / */
/*     DATA RMACH(2) / 577767777777777777776B / */
/*     DATA RMACH(3) / 377224000000000000000B / */
/*     DATA RMACH(4) / 377234000000000000000B / */
/*     DATA RMACH(5) / 377774642023241175720B / */

/*     MACHINE CONSTANTS FOR THE DATA GENERAL ECLIPSE S/200 */
/*     NOTE - IT MAY BE APPROPRIATE TO INCLUDE THE FOLLOWING CARD - */
/*     STATIC RMACH(5) */

/*     DATA SMALL /    20K,       0 / */
/*     DATA LARGE / 77777K, 177777K / */
/*     DATA RIGHT / 35420K,       0 / */
/*     DATA DIVER / 36020K,       0 / */
/*     DATA LOG10 / 40423K,  42023K / */

/*     MACHINE CONSTANTS FOR THE DEC ALPHA */
/*     USING G_FLOAT */

/*     DATA RMACH(1) / '00000080'X / */
/*     DATA RMACH(2) / 'FFFF7FFF'X / */
/*     DATA RMACH(3) / '00003480'X / */
/*     DATA RMACH(4) / '00003500'X / */
/*     DATA RMACH(5) / '209B3F9A'X / */

/*     MACHINE CONSTANTS FOR THE DEC ALPHA */
/*     USING IEEE_FLOAT */

/*     DATA RMACH(1) / '00800000'X / */
/*     DATA RMACH(2) / '7F7FFFFF'X / */
/*     DATA RMACH(3) / '33800000'X / */
/*     DATA RMACH(4) / '34000000'X / */
/*     DATA RMACH(5) / '3E9A209B'X / */

/*     MACHINE CONSTANTS FOR THE DEC RISC */

/*     DATA RMACH(1) / Z'00800000' / */
/*     DATA RMACH(2) / Z'7F7FFFFF' / */
/*     DATA RMACH(3) / Z'33800000' / */
/*     DATA RMACH(4) / Z'34000000' / */
/*     DATA RMACH(5) / Z'3E9A209B' / */

/*     MACHINE CONSTANTS FOR THE DEC VAX */
/*     (EXPRESSED IN INTEGER AND HEXADECIMAL) */
/*     THE HEX FORMAT BELOW MAY NOT BE SUITABLE FOR UNIX SYSTEMS */
/*     THE INTEGER FORMAT SHOULD BE OK FOR UNIX SYSTEMS */

/*     DATA SMALL(1) /       128 / */
/*     DATA LARGE(1) /    -32769 / */
/*     DATA RIGHT(1) /     13440 / */
/*     DATA DIVER(1) /     13568 / */
/*     DATA LOG10(1) / 547045274 / */

/*     DATA SMALL(1) / Z00000080 / */
/*     DATA LARGE(1) / ZFFFF7FFF / */
/*     DATA RIGHT(1) / Z00003480 / */
/*     DATA DIVER(1) / Z00003500 / */
/*     DATA LOG10(1) / Z209B3F9A / */

/*     MACHINE CONSTANTS FOR THE ELXSI 6400 */
/*     (ASSUMING REAL*4 IS THE DEFAULT REAL) */

/*     DATA SMALL(1) / '00800000'X / */
/*     DATA LARGE(1) / '7F7FFFFF'X / */
/*     DATA RIGHT(1) / '33800000'X / */
/*     DATA DIVER(1) / '34000000'X / */
/*     DATA LOG10(1) / '3E9A209B'X / */

/*     MACHINE CONSTANTS FOR THE HARRIS 220 */

/*     DATA SMALL(1), SMALL(2) / '20000000, '00000201 / */
/*     DATA LARGE(1), LARGE(2) / '37777777, '00000177 / */
/*     DATA RIGHT(1), RIGHT(2) / '20000000, '00000352 / */
/*     DATA DIVER(1), DIVER(2) / '20000000, '00000353 / */
/*     DATA LOG10(1), LOG10(2) / '23210115, '00000377 / */

/*     MACHINE CONSTANTS FOR THE HONEYWELL 600/6000 SERIES */

/*     DATA RMACH(1) / O402400000000 / */
/*     DATA RMACH(2) / O376777777777 / */
/*     DATA RMACH(3) / O714400000000 / */
/*     DATA RMACH(4) / O716400000000 / */
/*     DATA RMACH(5) / O776464202324 / */

/*     MACHINE CONSTANTS FOR THE HP 730 */

/*     DATA RMACH(1) / Z'00800000' / */
/*     DATA RMACH(2) / Z'7F7FFFFF' / */
/*     DATA RMACH(3) / Z'33800000' / */
/*     DATA RMACH(4) / Z'34000000' / */
/*     DATA RMACH(5) / Z'3E9A209B' / */

/*     MACHINE CONSTANTS FOR THE HP 2100 */
/*     3 WORD DOUBLE PRECISION WITH FTN4 */

/*     DATA SMALL(1), SMALL(2) / 40000B,       1 / */
/*     DATA LARGE(1), LARGE(2) / 77777B, 177776B / */
/*     DATA RIGHT(1), RIGHT(2) / 40000B,    325B / */
/*     DATA DIVER(1), DIVER(2) / 40000B,    327B / */
/*     DATA LOG10(1), LOG10(2) / 46420B,  46777B / */

/*     MACHINE CONSTANTS FOR THE HP 2100 */
/*     4 WORD DOUBLE PRECISION WITH FTN4 */

/*     DATA SMALL(1), SMALL(2) / 40000B,       1 / */
/*     DATA LARGE(1), LARGE(2) / 77777B, 177776B / */
/*     DATA RIGHT(1), RIGHT(2) / 40000B,    325B / */
/*     DATA DIVER(1), DIVER(2) / 40000B,    327B / */
/*     DATA LOG10(1), LOG10(2) / 46420B,  46777B / */

/*     MACHINE CONSTANTS FOR THE HP 9000 */

/*     DATA SMALL(1) / 00004000000B / */
/*     DATA LARGE(1) / 17677777777B / */
/*     DATA RIGHT(1) / 06340000000B / */
/*     DATA DIVER(1) / 06400000000B / */
/*     DATA LOG10(1) / 07646420233B / */

/*     MACHINE CONSTANTS FOR THE IBM 360/370 SERIES, */
/*     THE XEROX SIGMA 5/7/9, THE SEL SYSTEMS 85/86  AND */
/*     THE PERKIN ELMER (INTERDATA) 7/32. */

/*     DATA RMACH(1) / Z00100000 / */
/*     DATA RMACH(2) / Z7FFFFFFF / */
/*     DATA RMACH(3) / Z3B100000 / */
/*     DATA RMACH(4) / Z3C100000 / */
/*     DATA RMACH(5) / Z41134413 / */

/*     MACHINE CONSTANTS FOR THE IBM PC */

/*     DATA SMALL(1) / 1.18E-38      / */
/*     DATA LARGE(1) / 3.40E+38      / */
/*     DATA RIGHT(1) / 0.595E-07     / */
/*     DATA DIVER(1) / 1.19E-07      / */
/*     DATA LOG10(1) / 0.30102999566 / */

/*     MACHINE CONSTANTS FOR THE IBM RS 6000 */

/*     DATA RMACH(1) / Z'00800000' / */
/*     DATA RMACH(2) / Z'7F7FFFFF' / */
/*     DATA RMACH(3) / Z'33800000' / */
/*     DATA RMACH(4) / Z'34000000' / */
/*     DATA RMACH(5) / Z'3E9A209B' / */

/*     MACHINE CONSTANTS FOR THE INTEL i860 */

/*     DATA RMACH(1) / Z'00800000' / */
/*     DATA RMACH(2) / Z'7F7FFFFF' / */
/*     DATA RMACH(3) / Z'33800000' / */
/*     DATA RMACH(4) / Z'34000000' / */
/*     DATA RMACH(5) / Z'3E9A209B' / */

/*     MACHINE CONSTANTS FOR THE PDP-10 (KA OR KI PROCESSOR) */

/*     DATA RMACH(1) / "000400000000 / */
/*     DATA RMACH(2) / "377777777777 / */
/*     DATA RMACH(3) / "146400000000 / */
/*     DATA RMACH(4) / "147400000000 / */
/*     DATA RMACH(5) / "177464202324 / */

/*     MACHINE CONSTANTS FOR PDP-11 FORTRAN SUPPORTING */
/*     32-BIT INTEGERS (EXPRESSED IN INTEGER AND OCTAL). */

/*     DATA SMALL(1) /    8388608 / */
/*     DATA LARGE(1) / 2147483647 / */
/*     DATA RIGHT(1) /  880803840 / */
/*     DATA DIVER(1) /  889192448 / */
/*     DATA LOG10(1) / 1067065499 / */

/*     DATA RMACH(1) / O00040000000 / */
/*     DATA RMACH(2) / O17777777777 / */
/*     DATA RMACH(3) / O06440000000 / */
/*     DATA RMACH(4) / O06500000000 / */
/*     DATA RMACH(5) / O07746420233 / */

/*     MACHINE CONSTANTS FOR PDP-11 FORTRAN SUPPORTING */
/*     16-BIT INTEGERS  (EXPRESSED IN INTEGER AND OCTAL). */

/*     DATA SMALL(1), SMALL(2) /   128,     0 / */
/*     DATA LARGE(1), LARGE(2) / 32767,    -1 / */
/*     DATA RIGHT(1), RIGHT(2) / 13440,     0 / */
/*     DATA DIVER(1), DIVER(2) / 13568,     0 / */
/*     DATA LOG10(1), LOG10(2) / 16282,  8347 / */

/*     DATA SMALL(1), SMALL(2) / O000200, O000000 / */
/*     DATA LARGE(1), LARGE(2) / O077777, O177777 / */
/*     DATA RIGHT(1), RIGHT(2) / O032200, O000000 / */
/*     DATA DIVER(1), DIVER(2) / O032400, O000000 / */
/*     DATA LOG10(1), LOG10(2) / O037632, O020233 / */

/*     MACHINE CONSTANTS FOR THE SILICON GRAPHICS */

/*     DATA RMACH(1) / Z'00800000' / */
/*     DATA RMACH(2) / Z'7F7FFFFF' / */
/*     DATA RMACH(3) / Z'33800000' / */
/*     DATA RMACH(4) / Z'34000000' / */
/*     DATA RMACH(5) / Z'3E9A209B' / */

/*     MACHINE CONSTANTS FOR THE SUN */

/*     DATA RMACH(1) / Z'00800000' / */
/*     DATA RMACH(2) / Z'7F7FFFFF' / */
/*     DATA RMACH(3) / Z'33800000' / */
/*     DATA RMACH(4) / Z'34000000' / */
/*     DATA RMACH(5) / Z'3E9A209B' / */

/*     MACHINE CONSTANTS FOR THE SUN */
/*     USING THE -r8 COMPILER OPTION */

/*     DATA RMACH(1) / Z'0010000000000000' / */
/*     DATA RMACH(2) / Z'7FEFFFFFFFFFFFFF' / */
/*     DATA RMACH(3) / Z'3CA0000000000000' / */
/*     DATA RMACH(4) / Z'3CB0000000000000' / */
/*     DATA RMACH(5) / Z'3FD34413509F79FF' / */

/*     MACHINE CONSTANTS FOR THE UNIVAC 1100 SERIES */

/*     DATA RMACH(1) / O000400000000 / */
/*     DATA RMACH(2) / O377777777777 / */
/*     DATA RMACH(3) / O146400000000 / */
/*     DATA RMACH(4) / O147400000000 / */
/*     DATA RMACH(5) / O177464202324 / */

/*     MACHINE CONSTANTS FOR THE Z80 MICROPROCESSOR */

/*     DATA SMALL(1), SMALL(2) /     0,    256/ */
/*     DATA LARGE(1), LARGE(2) /    -1,   -129/ */
/*     DATA RIGHT(1), RIGHT(2) /     0,  26880/ */
/*     DATA DIVER(1), DIVER(2) /     0,  27136/ */
/*     DATA LOG10(1), LOG10(2) /  8347,  32538/ */

/* ***FIRST EXECUTABLE STATEMENT  R1MACH */
    if (*i__ < 1 || *i__ > 5) {
	xermsg_("SLATEC", "R1MACH", "I OUT OF BOUNDS", &c__1, &c__2, (ftnlen)
		6, (ftnlen)6, (ftnlen)15);
    }

    ret_val = rmach[*i__ - 1];
    return ret_val;

} /* r1mach_ */


