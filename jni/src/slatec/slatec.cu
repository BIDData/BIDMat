#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "MatKernel.hpp"

#include "gutils.cu"
#include "gyermsg.cu"
#include "gr1mach.cu"
#include "gi1mach.cu"
#include "gcsevl.cu"
#include "ginits.cu"
#include "gcot.cu"
#include "gpsi.cu"
#include "gpsifn.cu"

void ssetsizes(long long N, dim3 *gridp, int *nthreadsp) {
  int nblocks = 1;
  int nthreads = 32;
  int threads_per_block = 512;
  while (1L * nblocks * nthreads < N) {
    if (nblocks < 16) {
      nblocks = 2*nblocks;
    } else if (nthreads < threads_per_block) {
      nthreads = 2*nthreads;
    } else {
      nblocks = 2*nblocks;
    }
  }
  gridp->y = 1 + (nblocks-1)/65536;
  gridp->x = 1 + (nblocks-1)/gridp->y;
  gridp->z = 1;
  *nthreadsp = nthreads;
}

__device__ float fn_psi(float a) {return slatec_psi(&a);}

__device__ float fn_psifn(float a, float n) {
  float ans; long nn = (long)n, m = 1, ierr, nz;
  slatec_psifn(&a, &nn, &m, &m, &ans, &nz, &ierr);
  if (nn % 2 == 0) ans = - ans;
  return ans/tgammaf(n+1);
}

__device__ float fn_psiinv(float a) {
  float x;
  long i, c0 = 0, kode = 1, cn = 2, ierr, nz;
  float bb[2];
  if (a >= -2.2f) {
    x = expf(a) + 0.5f;
  } else {
    x = -1/(a + 0.5772156649f);
  }
  for (i = 0; i < 3; i++) {
    slatec_psifn(&x, &c0, &kode, &cn, bb, &nz, &ierr);
    x = x + (bb[0] + a)/bb[1];
  }
  return x;
}

__device__ const fntype slatec_gfctns[] = {
    fn_psi,
    fn_psiinv
};

__device__ const optype slatec_gfctns2[] = {
    fn_psifn,
};

__global__ void __slatec_gfun(float *A, float *B, int N, int opn) {
  fntype fn = slatec_gfctns[opn];
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < N; i += blockDim.x * gridDim.x * gridDim.y) {
    B[i] = fn(A[i]);
  }
}

int slatec_gfun(float *A, float *B, int N, int opn) {
  int nthreads;
  dim3 griddims;
  ssetsizes(N, &griddims, &nthreads);
  __slatec_gfun<<<griddims,nthreads>>>(A, B, N, opn);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

__global__ void __slatec_gfun2(int nrows, int ncols, float *A, int ar, int ac, float *B, int br, int bc, float *C, int cc, int opn) {
  optype fn = slatec_gfctns2[opn];
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  int row, col;
  for (int i = ip; i < nrows*ncols; i += blockDim.x * gridDim.x * gridDim.y) {
    col = i / nrows;
    row = i - col * nrows;
    C[row+col*cc] = fn(A[row*ar+col*ac], B[row*br+col*bc]);
  }
}

int slatec_gfun2(int nrows, int ncols, float *A, int ar, int ac, float *B, int br, int bc, float *C, int cc, int opn) {
  int nthreads;
  dim3 griddims;
  ssetsizes(nrows*ncols, &griddims, &nthreads);
  __slatec_gfun2<<<griddims,nthreads>>>(nrows, ncols, A, ar, ac, B, br, bc, C, cc, opn);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

