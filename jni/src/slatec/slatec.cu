#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "MatKernel.hpp"

#include "gutils.cu"
#include "gyermsg.cu"
#include "gr1mach.cu"
#include "gi1mach.cu"
#include "gcsevl.cu"
#include "ginits.cu"
#include "gcot.cu"
#include "gpsi.cu"
#include "gpsifn.cu"

void ssetsizes(long long N, dim3 *gridp, int *nthreadsp) {
  int nblocks = 1;
  int nthreads = 32;
  int threads_per_block = 512;
  while (1L * nblocks * nthreads < N) {
    if (nblocks < 16) {
      nblocks = 2*nblocks;
    } else if (nthreads < threads_per_block) {
      nthreads = 2*nthreads;
    } else {
      nblocks = 2*nblocks;
    }
  }
  gridp->y = 1 + (nblocks-1)/65536;
  gridp->x = 1 + (nblocks-1)/gridp->y;
  gridp->z = 1;
  *nthreadsp = nthreads;
}

__device__ float fn_psi(float a) {return slatec_psi(&a);}

__device__ float fn_psifn(float a, float n) {
  float ans; long nn = (long)n, m = 1, ierr, nz;
  slatec_psifn(&a, &nn, &m, &m, &ans, &nz, &ierr);
  if (nn % 2 == 0) ans = - ans;
  return ans/tgammaf(n+1);
}

__device__ float fn_psiinv(float a) {
  float x;
  long i, c0 = 0, kode = 1, cn = 2, ierr, nz;
  float bb[2];
  if (a >= -2.2f) {
    x = expf(a) + 0.5f;
  } else {
    x = -1/(a + 0.5772156649f);
  }
  for (i = 0; i < 3; i++) {
    slatec_psifn(&x, &c0, &kode, &cn, bb, &nz, &ierr);
    x = x + (bb[0] + a)/bb[1];
  }
  return x;
}

__device__ const fntype slatec_gfctns[] = {
    fn_psi,
    fn_psiinv
};

__global__ void __slatec_gfun(float *A, float *B, int N, int opn) {
  fntype fn = slatec_gfctns[opn];
  int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
  for (int i = ip; i < N; i += blockDim.x * gridDim.x * gridDim.y) {
    B[i] = fn(A[i]);
  }
}

int slatec_gfun(float *A, float *B, int N, int opn) {
  int nthreads;
  dim3 griddims;
  ssetsizes(N, &griddims, &nthreads);
  __slatec_gfun<<<griddims,nthreads>>>(A, B, N, opn);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

