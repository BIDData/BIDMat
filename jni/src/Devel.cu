#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "MatKernel.hpp"

#ifdef __CUDA_ARCH__ 
#if __CUDA_ARCH__ > 200

#define edcellupdate(RR,RP1,RP2,RPP,WUN,TMP)                                                               \
  asm("vmin4.s32.s32.s32.add" "%0, %1.b3210, %2.b4321, %3;": "=r" (RR) : "r" (RP1), "r" (RP2), "r" (WUN)); \
  asm("vadd4.s32.s32.s32" "%0, %1, %2, %3;": "=r" (TMP) : "r" (MM), "r" (RZ), "r" (RR));                   \
  asm("vmin4.s32.s32.s32" "%0, %1, %2, %3;": "=r" (RR) : "r" (TMP), "r" (RR), "r" (RR));       

#define hammingcell(A0,A1,B0,W0,C,TMP,TMP1,ZERO)                                                          \
  asm("and.b32" "%0, %1, %2;": "=r" (TMP) : "r" (A0), "r" (B0));                                          \
  asm("vset4.s32.s32.eq" "%0, %1, %2, %3;": "=r" (TMP) : "r" (TMP), "r" (ZERO), "r" (ZERO));              \
  asm("vsub4.s32.s32.s32" "%0, %1, %2, %3;": "=r" (TMP1) : "r" (ZERO), "r" (TMP), "r" (ZERO));            \
  asm("vmin4.u32.u32.u32.add" "%0, %1, %2, %3;": "=r" (C) : "r" (W0), "r" (TMP1), "r" (C));               \
  asm("vmax4.u32.u32.u32" "%0, %1.b4321, %2.b4321, %3;": "=r" (A0) : "r" (A0), "r" (A1), "r" (ZERO));  


#define hammingcell2(A0,A1,B0,W0,C0,TMP0,A2,A3,B1,W1,C1,TMP1,ZERO)                                        \
  asm("and.b32" "%0, %1, %2;": "=r" (TMP0) : "r" (A0), "r" (B0));                                         \
  asm("and.b32" "%0, %1, %2;": "=r" (TMP1) : "r" (A2), "r" (B1));                                         \
  asm("vset4.s32.s32.eq" "%0, %1, %2, %3;": "=r" (TMP0) : "r" (TMP0), "r" (ZERO), "r" (ZERO));            \
  asm("vset4.s32.s32.eq" "%0, %1, %2, %3;": "=r" (TMP1) : "r" (TMP1), "r" (ZERO), "r" (ZERO));            \
  asm("vsub4.s32.s32.s32" "%0, %1, %2, %3;": "=r" (TMP0) : "r" (ZERO), "r" (TMP0), "r" (ZERO));           \
  asm("vsub4.s32.s32.s32" "%0, %1, %2, %3;": "=r" (TMP1) : "r" (ZERO), "r" (TMP1), "r" (ZERO));           \
  asm("vmin4.u32.u32.u32.add" "%0, %1, %2, %3;": "=r" (C0) : "r" (W0), "r" (TMP0), "r" (C0));             \
  asm("vmin4.u32.u32.u32.add" "%0, %1, %2, %3;": "=r" (C1) : "r" (W1), "r" (TMP1), "r" (C1));             \
  asm("vmax4.u32.u32.u32" "%0, %1.b4321, %2.b4321, %3;": "=r" (A0) : "r" (A0), "r" (A1), "r" (ZERO));     \
  asm("vmax4.u32.u32.u32" "%0, %1.b4321, %2.b4321, %3;": "=r" (A2) : "r" (A2), "r" (A3), "r" (ZERO)); 

#define hammingend(A0)                               \
  asm("shr.b32" "%0, %1, 8;": "=r" (A0) : "r" (A0)); 

template<int VECLEN, int NVEC, int TLEN>
  __global__ void __hammingdists(int *a, int *b, int *w, int *op, int *ow, int n) {   
  __shared__ int sa[TLEN];
  __shared__ int sb[32][VECLEN*NVEC+1];
  __shared__ int sw[32][VECLEN*NVEC+1];
  __shared__ int sop[32];
  __shared__ int sow[32];
  register int aa[VECLEN+1];           
  register int bb[VECLEN];
  register int ww[VECLEN];
  int i, ioff, ioffmv, ip, tmp, tmp1, j, k, c, cmin, imin;
  int zero = 0;
  int sid = threadIdx.x + blockDim.x * threadIdx.y;

  if (threadIdx.y + blockDim.y * blockIdx.x < n) {

    // Load data into shared memory
    for (i = 0; i < TLEN/1024; i++) {
      sa[sid + i*1024] = a[sid + i*1024 + TLEN*blockIdx.x];
    }
    for (i = 0; i < VECLEN*NVEC/32; i++) {
      sb[threadIdx.y][threadIdx.x + i*blockDim.x] = b[sid + i*1024 + VECLEN*NVEC*blockIdx.x];
      sw[threadIdx.y][threadIdx.x + i*blockDim.x] = w[sid + i*1024 + VECLEN*NVEC*blockIdx.x];
    }
    __syncthreads();

    ip = threadIdx.x / NVEC;
    ioffmv = (threadIdx.x % NVEC) * VECLEN;
    ioff = ioffmv + ip * (TLEN*NVEC/32);
    cmin = 0x7fffffff;
    imin = -1;

    // Load data for this thread into registers
#pragma unroll
    for (j = 0; j < VECLEN; j++) {
      tmp = j + ioff;
      if (tmp < TLEN) {
        aa[j] = sa[tmp];
      }
      bb[j] = sb[threadIdx.y][j + ioffmv];
      ww[j] = sw[threadIdx.y][j + ioffmv];
    }
    // Step through offsets in A string
    for (j = 0; j < TLEN*NVEC/8; j++) {
      tmp = VECLEN + ioff + j / 4;
      if (tmp - ioffmv < TLEN - VECLEN * NVEC) {
        if (j % 4 == 0) {
          aa[VECLEN] = sa[tmp];
        }
        c = 0;
        // Inner loop over the length of the vector in registers
#pragma unroll
        for (k = 0; k < VECLEN; k++) {
          tmp1 = aa[k];
          hammingcell(tmp1, aa[k+1], bb[k], ww[k], c, tmp, tmp, zero);
          aa[k] = tmp1;
        }
        tmp1 = aa[VECLEN];
        hammingend(tmp1);
        aa[VECLEN] = tmp1;
        // Need to sum over NVEC to get complete score for a string
#pragma unroll
        for (k = 1; k < NVEC; k *= 2) {    
          tmp = __shfl_down(c, k);  
          c = c + tmp;
        }
        // Now compare with the accumulated min
        if (c < cmin) {
          cmin = c;
          imin = 4 * ioff + j;
        }
      }
    }
    // Compute the min across groups of NVEC threads in this warp
    for (k = NVEC; k < 32; k *= 2) {    
      tmp = __shfl_down(cmin, k);
      tmp1 = __shfl_down(imin, k);
      if (tmp < cmin) {
        cmin = tmp;
        imin = tmp1;
      }
    }
    // Save to shared memory in prep for saving to main memory
    if (threadIdx.x == 0) {
      sop[threadIdx.y] = imin;
      sow[threadIdx.y] = cmin;
    }
    __syncthreads();
    // Save to main memory
    if (threadIdx.y == 0) {
      op[threadIdx.x + 32*blockIdx.x] = sop[threadIdx.x];
      ow[threadIdx.x + 32*blockIdx.x] = sow[threadIdx.x];
    }
  }
}

__global__ void __veccmp(int *a, int *b, int *d) {
  int xa = *a;
  int xb = *b;
  int xc = 0;
  int xd = 0;
  asm("vset4.s32.s32.ne" "%0, %1.b0000, %2, %3;": "=r" (xd) : "r" (xa), "r" (xb), "r" (xc));
  *d++ = xd;
  asm("vset4.s32.s32.ne" "%0, %1.b1111, %2, %3;": "=r" (xd) : "r" (xa), "r" (xb), "r" (xc));
  *d++ = xd;
  asm("vset4.s32.s32.ne" "%0, %1.b2222, %2, %3;": "=r" (xd) : "r" (xa), "r" (xb), "r" (xc));
  *d++ = xd;
  asm("vset4.s32.s32.ne" "%0, %1.b3333, %2, %3;": "=r" (xd) : "r" (xa), "r" (xb), "r" (xc));
  *d = xd;
}
#else
__global__ void __veccmp(int *a, int *b, int *d) {
  printf("__veccmp() not defined for CUDA Arch < 300\n");
}

template<int VECLEN, int NVEC, int TLEN>
__global__ void __hammingdists(int *a, int *b, int *w, int *op, int *ow, int n) {
  printf("__hammingdists() not defined for CUDA Arch < 300\n");
}
#endif
#else
__global__ void __veccmp(int *a, int *b, int *d) {
  printf("__veccmp() not defined for CUDA Arch < 300\n");
}

template<int VECLEN, int NVEC, int TLEN>
__global__ void __hammingdists(int *a, int *b, int *w, int *op, int *ow, int n) {
  printf("__hammingdists() not defined for CUDA Arch < 300\n");
}
#endif

int veccmp(int *a, int *b, int *d) {
  __veccmp<<<1,1>>>(a, b, d);
  return 0;
}

int hammingdists(int *a, int *b, int *w, int *op, int *ow, int n) {    
  int nb = 1+((n-1)/32);
  dim3 blockdims(32,32,1);
  __hammingdists<8,4,1024><<<nb,blockdims>>>(a, b, w, op, ow, n);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}    
