#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#ifdef __CUDA_ARCH__ 
#if __CUDA_ARCH__ > 200

// Compute one level of random forest evaluation for a set of 32 trees. 
// The input is a dense feature matrix (feats) which is nrows (features) by ncols (samples).
//   ns is the number of random features used for each tree node.
//   tstride is the tree stride, i.e. how far to step in the trees array to access data from the next tree. 
//   ntrees is the number of trees (can be less than 32 but this wont be efficient in that case). 
//   trees is an array containing the feature indices. It is an ntrees x tstride matrix
//   tpos is an ntrees x ncols matrix containing the position indices for the parent nodes in the trees.
//   i.e. tpos indicates where each sample is in the traversal up to this depth in the trees. 
//   otpos is the output, which is an index for one of two child nodes for each parent, based on whether
//   the current feature sample sum is greater than the threshold.

// The trees array is really nnodes x ns x ntrees (i.e. tstride = nnodes x ns), where nnodes is the number of
// nodes in a single tree at the current depth. 
//  
// In each column of ns feature indices for a tree node, the 0^th index is actually the floating point threshold for the node. 
// It is converted and saved in a variable named fthresh

// currently the number of rows for the trees array must be a multiple of 32. 

template<int ATHREADS, int BTHREADS, int REPTREES>
__global__ void __treeprod(unsigned int *trees, float *feats, int *tpos, float *otv, int nrows, int ncols, int ns, int tstride, int ntrees) {

  int bd;
  __shared__ int pos[REPTREES][ATHREADS];
  __shared__ float totals[REPTREES][ATHREADS];
  unsigned int tind;
  float vv[REPTREES];

  for (bd = blockIdx.x; bd < ncols; bd += gridDim.x) {
    // Read in the index of parent for each tree
    if (threadIdx.x + threadIdx.y*ATHREADS < ntrees) {
      pos[threadIdx.y][threadIdx.x] = tpos[threadIdx.x + threadIdx.y*ATHREADS + ntrees * bd];
    }

    // Now read the tree node vectors associated with these trees
    __syncthreads();
#pragma unroll
    for (int k = 0; k < REPTREES; k++) {
      vv[k] = 0;
      if (threadIdx.y + k*BTHREADS < ntrees) {
        for (int j = threadIdx.x; j < ns+1; j += blockDim.x) {
          tind = trees[j + (ns+1)*pos[k][threadIdx.y] + (threadIdx.y+k*BTHREADS)*tstride];
          if (j > 0) {
            vv[k] += feats[tind + bd * nrows];  
          }              
        }
      }
    }

    // Sum the contents of the totals array
#pragma unroll
    for (int i = 1; i < 32; i *= 2) {
#pragma unroll
      for (int k = 0; k < REPTREES; k++) {
        vv[k] += __shfl_down(vv[k], i);
      }
    }
    if (threadIdx.x == 0) {
#pragma unroll
      for (int k = 0; k < REPTREES; k++) {   // and save in the totals array
        totals[k][threadIdx.y] = vv[k];
      }
    }

    // save
    __syncthreads();
    if (threadIdx.x + threadIdx.y*ATHREADS < ntrees) {
      otv[threadIdx.x + threadIdx.y*ATHREADS + ntrees * bd] = totals[threadIdx.y][threadIdx.x];
    }  
    __syncthreads();
  }
} 


template<int ATHREADS, int BTHREADS, int REPTREES>
__global__ void __treesteps(unsigned int *trees, float *feats, int *tpos, int *otpos, int nrows, int ncols, int ns, int tstride, int ntrees, int tdepth) {

  int newt, bd;
  __shared__ int pos[REPTREES][ATHREADS];
  __shared__ float thresh[REPTREES][ATHREADS];
  __shared__ float totals[REPTREES][ATHREADS];
  unsigned int tind;
  float vv[REPTREES];

  for (bd = blockIdx.x; bd < ncols; bd += gridDim.x) {
    // Read in the index of parent for each tree
    if (threadIdx.x + threadIdx.y*ATHREADS < ntrees) {
      pos[threadIdx.y][threadIdx.x] = tpos[threadIdx.x + threadIdx.y*ATHREADS + ntrees * bd];
    }
    for (int id = 0; id < tdepth; id ++) {
      // Now read the tree node vectors associated with these trees
      __syncthreads();
#pragma unroll
      for (int k = 0; k < REPTREES; k++) {
        vv[k] = 0;
        if (threadIdx.y + k*BTHREADS < ntrees) {
          for (int j = threadIdx.x; j < ns+1; j += blockDim.x) {
            tind = trees[j + (ns+1)*pos[k][threadIdx.y] + (threadIdx.y+k*BTHREADS)*tstride];
            if (j == 0) {
              thresh[k][threadIdx.y] = *((float *)&tind);
            } else {
              vv[k] += feats[tind + bd * nrows];  
            }              
          }
        }
      }

      // Sum the contents of the totals array
#pragma unroll
      for (int i = 1; i < 32; i *= 2) {
#pragma unroll
        for (int k = 0; k < REPTREES; k++) {
          vv[k] += __shfl_down(vv[k], i);
        }
      }
      if (threadIdx.x == 0) {
#pragma unroll
        for (int k = 0; k < REPTREES; k++) {   // and save in the totals array
          totals[k][threadIdx.y] = vv[k];
        }
      }

      // check thresholds and save as needed
      __syncthreads();
      if (threadIdx.x + threadIdx.y*ATHREADS < ntrees) {
        newt = 2 * pos[threadIdx.y][threadIdx.x] + 1;
        if (totals[threadIdx.y][threadIdx.x] > thresh[threadIdx.y][threadIdx.x]) {
          newt++;
        }
        pos[threadIdx.y][threadIdx.x] = newt; 
      }  
      __syncthreads();
    }
    if (threadIdx.x + threadIdx.y*ATHREADS < ntrees) {
      otpos[threadIdx.x + threadIdx.y*ATHREADS + ntrees * bd] = pos[threadIdx.y][threadIdx.x];
    }
  }
} 

#else
template<int ATHREADS, int BTHREADS, int REPTREES>
__global__ void __treeprod(unsigned int *trees, float *feats, int *tpos, float *otval, int nrows, int ncols, int ns, int tstride, int ntrees) {}
template<int ATHREADS, int BTHREADS, int REPTREES>
__global__ void __treesteps(unsigned int *trees, float *feats, int *tpos, int *otpos, int nrows, int ncols, int ns, int tstride, int ntrees, int tdepth) {}
#endif
#else
template<int ATHREADS, int BTHREADS, int REPTREES>
__global__ void __treeprod(unsigned int *trees, float *feats, int *tpos, float *otval, int nrows, int ncols, int ns, int tstride, int ntrees) {}
template<int ATHREADS, int BTHREADS, int REPTREES>
  __global__ void __treesteps(unsigned int *trees, float *feats, int *tpos, int *otpos, int nrows, int ncols, int ns, int tstride, int ntrees, int tdepth){}
#endif

int treeprod(unsigned int *trees, float *feats, int *tpos, float *otv, int nrows, int ncols, int ns, int tstride, int ntrees) {
  int nblks = min(1024, max(ncols/8, min(32, ncols)));
  dim3 blocks(32, 32, 1);
  int ntt;
  for (ntt = 32; ntt < ntrees; ntt *= 2) {}
  switch (ntt) {
  case (32) :
    __treeprod<32,32,1><<<nblks,blocks>>>(trees, feats, tpos, otv, nrows, ncols, ns, tstride, ntrees); break;
  case (64) :
    __treeprod<32,32,2><<<nblks,blocks>>>(trees, feats, tpos, otv, nrows, ncols, ns, tstride, ntrees); break;
  case (128) :
    __treeprod<32,32,4><<<nblks,blocks>>>(trees, feats, tpos, otv, nrows, ncols, ns, tstride, ntrees); break;
  case (256) :
    __treeprod<32,32,8><<<nblks,blocks>>>(trees, feats, tpos, otv, nrows, ncols, ns, tstride, ntrees); break;
  case (512) :
    __treeprod<32,32,16><<<nblks,blocks>>>(trees, feats, tpos, otv, nrows, ncols, ns, tstride, ntrees); break;
  case (1024) :
    __treeprod<32,32,32><<<nblks,blocks>>>(trees, feats, tpos, otv, nrows, ncols, ns, tstride, ntrees); break;
  } 
  hipDeviceSynchronize();
  int err = hipGetLastError();
  return err;
}


int treesteps(unsigned int *trees, float *feats, int *tpos, int *otpos, int nrows, int ncols, int ns, int tstride, int ntrees, int tdepth) {
  int nblks = min(1024, max(ncols/8, min(32, ncols)));
  dim3 blocks(32, 32, 1);
  int ntt;
  for (ntt = 32; ntt < ntrees; ntt *= 2) {}
  switch (ntt) {
  case (32) :
    __treesteps<32,32,1><<<nblks,blocks>>>(trees, feats, tpos, otpos, nrows, ncols, ns, tstride, ntrees, tdepth); break;
  case (64) :
    __treesteps<32,32,2><<<nblks,blocks>>>(trees, feats, tpos, otpos, nrows, ncols, ns, tstride, ntrees, tdepth); break;
  case (128) :
    __treesteps<32,32,4><<<nblks,blocks>>>(trees, feats, tpos, otpos, nrows, ncols, ns, tstride, ntrees, tdepth); break;
  case (256) :
    __treesteps<32,32,8><<<nblks,blocks>>>(trees, feats, tpos, otpos, nrows, ncols, ns, tstride, ntrees, tdepth); break;
  case (512) :
    __treesteps<32,32,16><<<nblks,blocks>>>(trees, feats, tpos, otpos, nrows, ncols, ns, tstride, ntrees, tdepth); break;
  case (1024) :
    __treesteps<32,32,32><<<nblks,blocks>>>(trees, feats, tpos, otpos, nrows, ncols, ns, tstride, ntrees, tdepth); break;
  } 
  hipDeviceSynchronize();
  int err = hipGetLastError();
  return err;
}

#define BLOCKDIM 32
const int INBLOCK = 4;

// copy and transpose columns of the input matrix into the output matrix. nrows refers to the input matrix 
// (and so is ncols for the output). ncols is the length of the iptrs array, which will be the number of 
// rows of the output matrix. iptrs specifies the columns of the input array to copy. 
// outstride is stride of the output matrix

__global__ void __icopy_transpose(int *iptrs, float *in, float *out, int outstride, int nrows, int ncols) {
  __shared__ float tile[BLOCKDIM][BLOCKDIM+1];
  int nx = BLOCKDIM * gridDim.x;
  int ny = BLOCKDIM * gridDim.y;
  int ix = BLOCKDIM * blockIdx.x;
  int iy = BLOCKDIM * blockIdx.y;

  for (int yb = iy; yb < ncols; yb += ny) {
    for (int xb = ix; xb < nrows; xb += nx) {
      if (xb + threadIdx.x < nrows) {
        int ylim = min(ncols, yb + BLOCKDIM);
        for (int y = threadIdx.y + yb; y < ylim; y += blockDim.y) {
          tile[threadIdx.x][y-yb] = in[threadIdx.x + xb + iptrs[y]*nrows];
        }
      }
      __syncthreads();
      if (yb + threadIdx.x < ncols) {
        int xlim = min(nrows, xb + BLOCKDIM);
        for (int x = threadIdx.y + xb; x < xlim; x += blockDim.y) {
          out[threadIdx.x + yb + x*outstride] = tile[x-xb][threadIdx.x];
        }
      }
      __syncthreads();
    }
  } 
}

int icopy_transpose(int *iptrs, float *in, float *out, int stride, int nrows, int ncols) {
  const dim3 griddims(20,256,1);
  const dim3 blockdims(BLOCKDIM,INBLOCK,1);
  hipError_t err;
  __icopy_transpose<<<griddims,blockdims>>>(iptrs, in, out, stride, nrows, ncols); 
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {fprintf(stderr, "cuda error in icopy_transpose"); return err;}
  return 0;
}

// copy and transpose the input matrix into columns of the output matrix. nrows, ncols refer to output matrix

__global__ void __ocopy_transpose(int *optrs, float *in, float *out, int instride, int nrows, int ncols) {
  int nx = BLOCKDIM * gridDim.x;
  int ny = BLOCKDIM * gridDim.y;
  int ix = BLOCKDIM * blockIdx.x;
  int iy = BLOCKDIM * blockIdx.y;
  __shared__ float tile[BLOCKDIM][BLOCKDIM+1];

  for (int yb = iy; yb < ncols; yb += ny) {
    for (int xb = ix; xb < nrows; xb += nx) {
      if (yb + threadIdx.x < ncols) {
        int xlim = min(nrows, xb + BLOCKDIM);
        for (int x = threadIdx.y + xb; x < xlim; x += blockDim.y) {
          tile[x-xb][threadIdx.x] = in[threadIdx.x + yb + x*instride];
        }
      }
      __syncthreads();
      if (xb + threadIdx.x < nrows) {
        int ylim = min(ncols, yb + BLOCKDIM);
        for (int y = threadIdx.y + yb; y < ylim; y += blockDim.y) {
          out[optrs[y]*nrows + threadIdx.x + xb] = tile[threadIdx.x][y-yb];
        }
      }
      __syncthreads();
    }
  } 
}

__global__ void __ocopy_transpose_add(int *optrs, float *in, float *out, int instride, int nrows, int ncols) {
  int nx = BLOCKDIM * gridDim.x;
  int ny = BLOCKDIM * gridDim.y;
  int ix = BLOCKDIM * blockIdx.x;
  int iy = BLOCKDIM * blockIdx.y;
  __shared__ float tile[BLOCKDIM][BLOCKDIM+1];

  for (int yb = iy; yb < ncols; yb += ny) {
    for (int xb = ix; xb < nrows; xb += nx) {
      if (yb + threadIdx.x < ncols) {
        int xlim = min(nrows, xb + BLOCKDIM);
        for (int x = threadIdx.y + xb; x < xlim; x += blockDim.y) {
          tile[x-xb][threadIdx.x] = in[threadIdx.x + yb + x*instride];
        }
      }
      __syncthreads();
      if (xb + threadIdx.x < nrows) {
        int ylim = min(ncols, yb + BLOCKDIM);
        for (int y = threadIdx.y + yb; y < ylim; y += blockDim.y) {
          atomicAdd(&out[optrs[y]*nrows + threadIdx.x + xb], tile[threadIdx.x][y-yb]);
        }
      }
      __syncthreads();
    }
  } 
}

__global__ void __ocopy_transpose_min(int *optrs, float *in, float *out, int instride, int nrows, int ncols) {
  int nx = BLOCKDIM * gridDim.x;
  int ny = BLOCKDIM * gridDim.y;
  int ix = BLOCKDIM * blockIdx.x;
  int iy = BLOCKDIM * blockIdx.y;
  __shared__ float tile[BLOCKDIM][BLOCKDIM+1];

  for (int yb = iy; yb < ncols; yb += ny) {
    for (int xb = ix; xb < nrows; xb += nx) {
      if (yb + threadIdx.x < ncols) {
        int xlim = min(nrows, xb + BLOCKDIM);
        for (int x = threadIdx.y + xb; x < xlim; x += blockDim.y) {
          tile[x-xb][threadIdx.x] = in[threadIdx.x + yb + x*instride];
        }
      }
      __syncthreads();
      if (xb + threadIdx.x < nrows) {
        int ylim = min(ncols, yb + BLOCKDIM);
        for (int y = threadIdx.y + yb; y < ylim; y += blockDim.y) {
          atomicMin((int *)&out[optrs[y]*nrows + threadIdx.x + xb], *(int *)(&tile[threadIdx.x][y-yb]));
        }
      }
      __syncthreads();
    }
  } 
}

int ocopy_transpose_add(int *optrs, float *in, float *out, int stride, int nrows, int ncols) {
  const dim3 griddims(20,256,1);
  const dim3 blockdims(BLOCKDIM,INBLOCK,1);
  hipError_t err;
  __ocopy_transpose_add<<<griddims,blockdims>>>(optrs, in, out, stride, nrows, ncols); 
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {fprintf(stderr, "cuda error in ocopy_transpose"); return err;}
  return 0;
}

int ocopy_transpose(int *optrs, float *in, float *out, int stride, int nrows, int ncols) {
  const dim3 griddims(20,256,1);
  const dim3 blockdims(BLOCKDIM,INBLOCK,1);
  hipError_t err;
  __ocopy_transpose<<<griddims,blockdims>>>(optrs, in, out, stride, nrows, ncols); 
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {fprintf(stderr, "cuda error in ocopy_transpose"); return err;}
  return 0;
}

int ocopy_transpose_min(int *optrs, float *in, float *out, int stride, int nrows, int ncols) {
  const dim3 griddims(20,256,1);
  const dim3 blockdims(BLOCKDIM,INBLOCK,1);
  hipError_t err;
  __ocopy_transpose_min<<<griddims,blockdims>>>(optrs, in, out, stride, nrows, ncols); 
  hipDeviceSynchronize();
  err = hipGetLastError();
  if (err != hipSuccess) {fprintf(stderr, "cuda error in ocopy_transpose"); return err;}
  return 0;
}


#ifdef __CUDA_ARCH__ 
#if __CUDA_ARCH__ > 200
__global__ void __cumsumi(int *in, int *out, int *jc, int nrows) {
  __shared__ int tots[32];
  int start = jc[blockIdx.x] + nrows * blockIdx.y;
  int end = jc[blockIdx.x+1] + nrows * blockIdx.y;
  int sum = 0;
  int tsum, tmp, ttot, ttot0;
  for (int i = start + threadIdx.x + threadIdx.y * blockDim.x; i < end; i += blockDim.x * blockDim.y) {
    tsum = in[i];
    tmp = __shfl_up(tsum, 1);
    if (threadIdx.x >= 1) tsum += tmp;
    tmp = __shfl_up(tsum, 2);
    if (threadIdx.x >= 2) tsum += tmp;
    tmp = __shfl_up(tsum, 4);
    if (threadIdx.x >= 4) tsum += tmp;
    tmp = __shfl_up(tsum, 8);
    if (threadIdx.x >= 8) tsum += tmp;
    tmp = __shfl_up(tsum, 16);
    if (threadIdx.x >= 16) tsum += tmp;
    ttot = __shfl(tsum, 31);
    ttot0 = ttot;
    __syncthreads();
    if (threadIdx.x == threadIdx.y) {
      tots[threadIdx.y] = ttot;
    }
    __syncthreads();
    for (int k = 1; k < blockDim.y; k *= 2) {
      if (threadIdx.y >= k) {
        if (threadIdx.x == threadIdx.y - k) {
          ttot += tots[threadIdx.x];
        }
      }
      __syncthreads();
      if (threadIdx.y >= k) {
        ttot = __shfl(ttot, threadIdx.y - k);
        if (threadIdx.x == threadIdx.y) {
          tots[threadIdx.y] = ttot;
        }
      }
      __syncthreads();
    }
    out[i] = sum + tsum + ttot - ttot0;
    if (threadIdx.x == blockDim.y - 1) {
      ttot = tots[threadIdx.x];
    }
    ttot = __shfl(ttot, blockDim.y  - 1);
    sum += ttot;
  }
}

__global__ void __maxs(float *in, float *out, int *outi, int *jc) {
  __shared__ float maxv[32];
  __shared__ int maxi[32];
  int start = jc[blockIdx.x];
  int end = jc[blockIdx.x+1];
  float vmax, vtmp;
  int imax, itmp, i, k;
  int istart = start + threadIdx.x + threadIdx.y * blockDim.x;

  if (istart < end) {
    vmax = in[istart];
    imax = istart;
  }

  for (i = istart + blockDim.x * blockDim.y; i < end; i += blockDim.x * blockDim.y) {
    vtmp = in[i];
    itmp = i;
    if (vtmp > vmax) {
      vmax = vtmp;
      imax = itmp;
    }
  }

  for (k = 1; k < blockDim.x; k *= 2) {
    vtmp = __shfl_up(vmax, k);
    itmp = __shfl_up(imax, k);
    if (threadIdx.x >= k) {
      if (vtmp > vmax) {
        vmax = vtmp;
        imax = itmp;
      }
    }
  }
  vmax = __shfl(vmax, blockDim.x - 1);
  imax = __shfl(imax, blockDim.x - 1);
  __syncthreads();

  if (threadIdx.x == threadIdx.y) {
    maxv[threadIdx.y] = vmax;
    maxi[threadIdx.y] = imax;
  }

  __syncthreads();
  if (threadIdx.y == 0) {
    vmax = maxv[threadIdx.x];
    imax = maxi[threadIdx.x];
  }
  __syncthreads();
  if (threadIdx.y == 0) {
    for (k = 1; k < blockDim.y; k *= 2) {
      vtmp = __shfl_up(vmax, k);
      itmp = __shfl_up(imax, k);
      if (threadIdx.x >= k) {
        if (vtmp > vmax) {
          vmax = vtmp;
          imax = itmp;
        }
      }
    }
    if (threadIdx.x == blockDim.y - 1) {
      out[blockIdx.x] = vmax;
      outi[blockIdx.x] = imax;
    }
  }
}
#else
__global__ void __cumsumi(int *in, int *out, int *jc, int nrows) {}
__global__ void __maxs(float *in, float *out, int *outi, int *jc) {}
#endif
#else
__global__ void __cumsumi(int *in, int *out, int *jc, int nrows) {}
__global__ void __maxs(float *in, float *out, int *outi, int *jc) {}
#endif

int cumsumi(int *in, int *out, int *jc, int nrows, int ncols, int m) {
  dim3 grid(m, ncols, 1);
  dim3 tblock(32, 32, 1);
  __cumsumi<<<grid,tblock>>>(in, out, jc, nrows);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}

int maxs(float *in, float *out, int *outi, int *jc, int m) {
  dim3 grid(m, 1, 1);
  dim3 tblock(32, 32, 1);
  __maxs<<<grid,tblock>>>(in, out, outi, jc);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  return err;
}
